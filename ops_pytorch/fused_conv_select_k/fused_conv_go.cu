#include "hip/hip_runtime.h"
// input: kernel_size(h,w), stride_size(h,w), distance(float), flag_padding, xyz(b,H,W,3), bhw_idx(b,H,W,3)
// output: selected_xyz(b, npoints, h*w, 3), selected_feature(b, npoints, h*w, 3)
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <cstdlib>        // Header file needed to use rand
#include "fused_conv_gpu.h"


__global__ void fused_conv_select_k_gpu(int batch_size, int H, int W, int npoints, int kernel_size_H,
                                        int kernel_size_W, int K, int flag, float distance, int stride_h,
                                        int stride_w, const float *xyz1,
                                        const float *xyz2, const int *idx_n2, const int *random_hw,
                                        long *selected_b_idx, long *selected_h_idx, long *selected_w_idx,
                                        float *valid_idx,
                                        float *valid_in_dis_idx, float *selected_mask, int small_h, int small_w) {

    int batch_index = blockIdx.x; //当前线程块索引
    int index_thread = threadIdx.x;
    int stride_thread = blockDim.x;

    int kernel_total = kernel_size_H * kernel_size_W;        // 一个kernel的大小
    int selected_W_idx = 0, selected_H_idx = 0;

    float dist_square = distance * distance;

    int kernel_half_H = kernel_size_H / 2;
    int kernel_half_W = kernel_size_W / 2;

    xyz1 += batch_index * H * W * 3;                //point cloud of current image
    xyz2 += batch_index * small_h * small_w * 3;
    idx_n2 += batch_index * npoints * 2;                                            // 2d coordinates of central points
    selected_b_idx += batch_index * npoints * K *
                      1; //(b, npoints, k, 1)，			// batch index of K selected points around central points
    selected_h_idx += batch_index * npoints * K * 1; //(b, npoints, k, 1)，
    selected_w_idx += batch_index * npoints * K * 1; //(b, npoints, k, 1)，

    valid_idx += batch_index * npoints * kernel_total *
                 1; //(b, npoints, h*w, 1)，	// coordinate-valid kernel points around central points
    valid_in_dis_idx += batch_index * npoints * kernel_total *
                        1; //(b, npoints, h*w, 1)， // distance-and-corrdinate-valid kernel points around central points

    selected_mask += batch_index * npoints * K * 1; //(b, npoints, h*w, 1)，坐标有效且距离有效的点，含复制的点（重复使用最近邻的点）


    //////////////      Fused  Conv  Between

    for (int current_n = index_thread; current_n < npoints; current_n += stride_thread)  //  output_W circle
    {

        int idx_w[500], idx_h[500];
        float Dist[500];

        for (int ii = 0; ii < 500; ++ii) {
            idx_w[ii] = 0;
            idx_h[ii] = 0;
            Dist[ii] = 1e10f;
        }

        int m_idx = 0;   //  mth point in each kernel
        int num_select = 0; // the number of selected points in each kernel
        int num_valid_idx = 0; // the number of valid points in each kernel

        selected_H_idx = idx_n2[current_n * 2 + 0];    // the  central points H idx of input 2d frame
        selected_W_idx = idx_n2[current_n * 2 + 1];    // the  central points W idx of input 2d frame

        float x_c = xyz1[selected_H_idx * W * 3 + selected_W_idx * 3 + 0];
        float y_c = xyz1[selected_H_idx * W * 3 + selected_W_idx * 3 + 1];
        float z_c = xyz1[selected_H_idx * W * 3 + selected_W_idx * 3 + 2];

        float Dist_c = max((x_c - 0) * (x_c - 0) + (y_c - 0) * (y_c - 0) + (z_c - 0) * (z_c - 0), 1e-10f);

        if (Dist_c <= 1e-10f)    //   not  valid  central  points of xyz1
        {
            continue;

        }

        //  valid  central  points of xyz2

        for (int current_HW_idx = 0;
             current_HW_idx < kernel_total; ++current_HW_idx) //select points in every kernel element
        {

            int kernel_HW_idx = random_hw[current_HW_idx]; //random_hw is random selected elements of kernel coordinations

            // stride is due to the points are in xyz2 which may be a down-sampled point cloud, the scale isn't the same as xyz1
            int kernel_select_H_idx =
                    selected_H_idx / stride_h + kernel_HW_idx / kernel_size_W - kernel_half_H; // random select ？？？
            int kernel_select_W_idx =
                    selected_W_idx / stride_w + kernel_HW_idx % kernel_size_W - kernel_half_W; // random select ？？？


            //TODO: change1
            if (flag & 0x2)  // bit boolean 0x2 == 0b0010
            {
                // only skip the point which is over the boundary along the H axis
                if ((kernel_select_H_idx < 0) ||
                    (kernel_select_H_idx >= small_h)) //  the region of padding points (not valid)
                {
                    ++m_idx;
                    continue;
                }
                // along the width axis, they should be treated as a circle shift
                if (kernel_select_W_idx < 0) {
                    kernel_select_W_idx = small_w + kernel_select_W_idx;   // circle shift
                }

                if (kernel_select_W_idx >= small_w) {
                    kernel_select_W_idx = kernel_select_W_idx - small_w;  // circle shift
                }
            }
            else{
                if ((kernel_select_H_idx < 0) || (kernel_select_H_idx >= small_h) || (kernel_select_W_idx < 0) ||
                    (kernel_select_W_idx >= small_w)) //  the region of padding points (not valid)
                {
                    ++m_idx;
                    continue;
                }

                //if (kernel_select_W_idx < 0)
                //{
                //	kernel_select_W_idx = small_w + kernel_select_W_idx;   ////    cylindrical project???
                //}

                //if (kernel_select_W_idx >= small_w)
                //{
                //	kernel_select_W_idx = kernel_select_W_idx - small_w;  ////    cylindrical project???
                //}
            }


            //  not the padding points

            float x_q = xyz2[kernel_select_H_idx * small_w * 3 + kernel_select_W_idx * 3 + 0];
            float y_q = xyz2[kernel_select_H_idx * small_w * 3 + kernel_select_W_idx * 3 + 1];
            float z_q = xyz2[kernel_select_H_idx * small_w * 3 + kernel_select_W_idx * 3 + 2];

            float Dist_q_0 = x_q * x_q + y_q * y_q + z_q * z_q;

            if (Dist_q_0 <= 1e-10f)  //  not valid xyz2 points
            {
                ++m_idx;
                continue;
            }

            // valid xyz2 points, calculate the distance

            //valid_idx[current_n * kernel_total * 1 + num_valid_idx * 1 + 0] = 1.0;//与有效点编号不一定对的上？
            //++num_valid_idx;

            float Dist_q = max((x_c - x_q) * (x_c - x_q) + (y_c - y_q) * (y_c - y_q) + (z_c - z_q) * (z_c - z_q),
                               1e-10f);

            if (Dist_q > dist_square)  // too far from the central points, regarding as not valid
            {
                ++m_idx;
                continue;
            }

            // selected_bhw_idx[current_n * K * 3 + m_idx * 3 + 0 ] = batch_index;
            // selected_bhw_idx[current_n * K * 3 + m_idx * 3 + 1 ] = kernel_select_H_idx;
            // selected_bhw_idx[current_n * K * 3 + m_idx * 3 + 2 ] = kernel_select_W_idx;
            // selected_mask[current_n * K * 1 + m_idx * 1 + 0 ] = 1.0;

            //valid_in_dis_idx[current_n * kernel_total * 1 + num_select * 1 + 0] = 1.0;

            Dist[m_idx] = Dist_q;
            idx_h[m_idx] = kernel_select_H_idx;
            idx_w[m_idx] = kernel_select_W_idx;

            ++m_idx;
            ++num_select;

            if (num_select >= kernel_total)  //  search all position
                break;

        }

        //?int sort_num = 0;

        for (int s_idx = 0; s_idx < K; ++s_idx)  // knn
        {
            int min_idx = s_idx;  // min_idx idx

            // find the min_idx
            for (int t = s_idx + 1; t < kernel_total; ++t) {
                if (Dist[t] < Dist[min_idx]) {
                    min_idx = t;
                }
            }

            // swap min_idx-th and i-th element
            if (min_idx != s_idx) {
                float tmp_dist = Dist[min_idx];
                int tmp_idx_w = idx_w[min_idx];
                int tmp_idx_h = idx_h[min_idx];

                Dist[min_idx] = Dist[s_idx];
                idx_w[min_idx] = idx_w[s_idx];
                idx_h[min_idx] = idx_h[s_idx];

                Dist[s_idx] = tmp_dist;
                idx_w[s_idx] = tmp_idx_w;
                idx_h[s_idx] = tmp_idx_h;

            }

            // TODO:change2, also a bit boolean 0b0001
            if ((flag & 0x1) && (s_idx == 0)) // copy the first selected point in xyz2 for K times
            {
                for (int k_idx = 0; k_idx < K; ++k_idx) {

                    selected_b_idx[current_n * K + k_idx] = batch_index;
                    selected_h_idx[current_n * K + k_idx] = idx_h[s_idx];
                    selected_w_idx[current_n * K + k_idx] = idx_w[s_idx];
                    selected_mask[current_n * K * 1 + k_idx * 1 + 0] = 1.0;

                }

            }     //  copy done


            if (Dist[s_idx] < 1e10f)  //  whether this is a valid points or not
            {

                selected_b_idx[current_n * K + s_idx] = batch_index;
                selected_h_idx[current_n * K + s_idx] = idx_h[s_idx];
                selected_w_idx[current_n * K + s_idx] = idx_w[s_idx];
                selected_mask[current_n * K * 1 + s_idx * 1 + 0] = 1.0;

            }

        }


    }

}


void FusedConvSelectKLauncher(int batch_size, int H, int W, int npoints, int kernel_size_H,
                              int kernel_size_W, int K, int flag, float distance, int stride_h, int stride_w,
                              const float *xyz1, const float *xyz2, const int *idx_n2, const int *random_hw,
                              long *selected_b_idx, long *selected_h_idx, long *selected_w_idx, float *valid_idx,
                              float *valid_in_dis_idx,
                              float *selected_mask, int small_h, int small_w, hipStream_t stream) {

    hipError_t err;

    fused_conv_select_k_gpu<<<batch_size, 512, 0, stream>>>(batch_size, H, W, npoints, kernel_size_H, kernel_size_W, K,
                                                            flag, distance, stride_h, stride_w, xyz1, xyz2, idx_n2,
                                                            random_hw, selected_b_idx, selected_h_idx, selected_w_idx,
                                                            valid_idx, valid_in_dis_idx, selected_mask, small_h,
                                                            small_w);

    //hipDeviceSynchronize();
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
